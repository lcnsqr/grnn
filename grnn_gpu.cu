#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include "pathio.h"
#include "grnn_gpu.h"

// Arquivo das amostras
#define TRAIN "train.bin"
// Arquivos de teste
#define TEST "test.bin"

// Obter a quantidade de cuda cores por multiprocessador a partir do Compute Capability
inline int _ConvertSMVer2Cores(int major, int minor) {
    // Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
    typedef struct {
        int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
        int Cores;
    } sSMtoCores;

    sSMtoCores nGpuArchCoresPerSM[] = {
        { 0x30, 192}, // Kepler Generation (SM 3.0) GK10x class
        { 0x32, 192}, // Kepler Generation (SM 3.2) GK10x class
        { 0x35, 192}, // Kepler Generation (SM 3.5) GK11x class
        { 0x37, 192}, // Kepler Generation (SM 3.7) GK21x class
        { 0x50, 128}, // Maxwell Generation (SM 5.0) GM10x class
        { 0x52, 128}, // Maxwell Generation (SM 5.2) GM20x class
        { 0x53, 128}, // Maxwell Generation (SM 5.3) GM20x class
        { 0x60, 64 }, // Pascal Generation (SM 6.0) GP100 class
        { 0x61, 128}, // Pascal Generation (SM 6.1) GP10x class
        { 0x62, 128}, // Pascal Generation (SM 6.2) GP10x class
        { 0x70, 64 }, // Volta Generation (SM 7.0) GV100 class

        {   -1, -1 }
    };

    int index = 0;

    while (nGpuArchCoresPerSM[index].SM != -1) {
        if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor))
        {
            return nGpuArchCoresPerSM[index].Cores;
        }

        index++;
    }

    // If we don't find the values, we default use the previous one to run properly
    printf("MapSMtoCores for SM %d.%d is undefined.  Default to use %d Cores/SM\n", major, minor, nGpuArchCoresPerSM[index-1].Cores);
    return nGpuArchCoresPerSM[index-1].Cores;
}

int main(int argc, char **argv){
	// Opções da linha de comando
	const char* outfile = NULL;
	float ss = 1;
	for(int i = 1; i < argc; i++){
		switch (argv[i][1]){
		case 'o':
			// Salvar resultado no arquivo indicado
			outfile = argv[i+1];
		break;
		case 's':
			// Escalar do parâmetro sigma
			ss = atof(argv[i+1]);
		break;
		}
	}

	// Identificar dispositivo
	init_gpu();

	// Cabeçalho do arquivo csv
	printf("Dispositivo\tGeração\tCapacidade\tMultiprocessadores\tCUDA Cores / MP\tMemória Global\tCUDA Driver\tCUDA Runtime\tDimensões da variável independente\tDimensões da variável dependente\tConjunto de treinamento\tConjunto de teste\tErro médio\n");

	// Hardware
	printf("%s\t", deviceProp.name);
	switch ( deviceProp.major ){
		case 3: printf("Kepler"); break;
		case 5: printf("Maxwell"); break;
		case 6: printf("Pascal"); break;
		case 7: printf("Volta"); break;
	}
	printf("\t");
	printf("%d.%d\t", deviceProp.major, deviceProp.minor);
	printf("%d\t", deviceProp.multiProcessorCount);
	printf("%d\t", _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor));
	printf("%.0f MB\t", (float)deviceProp.totalGlobalMem/1048576.0f);
	// Driver e Runtime
	int driverVersion = 0, runtimeVersion = 0;
	hipDriverGetVersion(&driverVersion);
	hipRuntimeGetVersion(&runtimeVersion);
	printf("%d.%d\t", driverVersion/1000, (driverVersion%100)/10);
	printf("%d.%d\t", runtimeVersion/1000, (runtimeVersion%100)/10);

	struct pathSet train, estim;
	// Carregar arquivo das amostras de treinamento
	pathSetLoad(TRAIN, &train);

	// Arquivo de teste
	pathSetLoad(TEST, &estim);

	printf("%d\t", train.dim[0]);
	printf("%d\t", train.dim[1]);
	printf("%d\t", train.total);
	printf("%d\t", estim.total);

	// Calcular o erro ou salvar um arquivo com o resultado
	// Soma dos erros das estimativas
	float errsum = 0;

	// Gerar estimativas
	estimar(&train, &estim, ss, &errsum);

	// Exibir erro médio
	printf("%f\n", errsum / (float)estim.total);

	// Salvar resultado no arquivo informado
	if (outfile != NULL ){
		pathSetSave(outfile, &estim);
	}

	return 0;
}
