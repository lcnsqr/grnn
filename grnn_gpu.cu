#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include "pathio.h"

// Arquivo das amostras
#define TRAIN "train.bin"
// Arquivos de teste
#define TEST "test.bin"

// Tamanho da memória compartilhada por bloco (GPU)
unsigned int sharedMemPerBlock;
// Número máximo de threads por bloco
unsigned int maxThreadsPerBlock;

// Implementação da distância entre dois vetores
float dist(float *v, float *w, int n){
	// Quadrado da distância euclidiana entre o vetores v e w
	float d = 0;
	for (int i = 0; i < n; i++){
		d += pow(w[i]-v[i], 2);
	}
	return d;
}

// Estimar a variável dependente (kernel)
// train: Conjunto de treinamento
// total: Total de amostras no conjunto de treinamento
// threadsPerBlock: Threads por bloco
// dim: Dimensões da variável independente e dependente
// x: Variável independente lida
// yPart: Acumulador das somas parciais
// s: Parâmetro da regressão
__global__ void estimKernel(float *train, const unsigned int total, const unsigned int threadsPerBlock, unsigned int *dim, float *x, float *yPart, const float s){
	// Ignorar se thread atual ultrapassou total de amostras
	if ( blockIdx.x * threadsPerBlock + threadIdx.x + 1 > total ) return;
	// Tamanho de cada amostra (variável independente e dependente)
	unsigned int dims = dim[0]+dim[1];
	// Fator comum das operações
	float f;
	// Distância estimando-amostra
	float d = 0;

	// Atalhos para a amostra na memória compartilhada
	float *sx;
	float *sy;

	// Carregar amostra na memória compartilhada
	extern __shared__ float sData[];
	sx = &sData[threadIdx.x * dims];
	sy = &sData[threadIdx.x * dims + dim[0]];

	// Guardar diferença entre estimando 
	// e variável independente da amostra
	float dif;

	for(int c = 0; c < dim[0]; c++){
		sx[c] = train[(blockIdx.x * threadsPerBlock + threadIdx.x) * dims + c];
		// Distância entre estimando e variável independente da amostra
		//d += pow( x[c] - sx[c], 2);
		dif = __fsub_rn(x[c], sx[c]);
		d = __fadd_rn(d, __fmul_rn(dif, dif));
	}
	for(int c = 0; c < dim[1]; c++){
		sy[c] = train[(blockIdx.x * threadsPerBlock + threadIdx.x) * dims + dim[0] + c];
	}

	// Sincronizar threads
	__syncthreads();
	
	// Fator comum
	//f = exp( -d / s );
	f = __expf( __fdiv_rn(-d, s));

	// Atalhos para soma parcial
	float *numer = &yPart[blockIdx.x * 2 * dim[1]];
	float *denom = &yPart[blockIdx.x * 2 * dim[1] + dim[1]];
	// Efetuar a soma parcial para cada dimensão da variável dependente
	for(unsigned int c = 0; c < dim[1]; c++){
		// Parcial do numerador
		atomicAdd( &numer[c], sy[c] * f );
		// Parcial do denominador
		atomicAdd( &denom[c], f );
	}
}

void testarDev(struct pathSet *train, struct pathSet *test, float sigma, float *errsum){
	// Registrar conjunto de treinamento na memória para 
	// evitar paginação e agilizar o acesso pela GPU ao
	// mapear a memória entre o host e a memória da GPU
	hipHostRegister(train->data.f, train->size, hipHostRegisterMapped);
	// Copiar conjunto de treinamento para memória do dispositivo
	float *trainDataDev;
	//hipHostGetDevicePointer(&trainDataDev, train->data.f, 0);
	hipMalloc(&trainDataDev, train->size);
	hipMemcpy(trainDataDev, train->data.f, train->size, hipMemcpyHostToDevice);

	// Dimensões da variável independente e dependente
	unsigned int *dim;
	unsigned int *dimDev;
	dim = train->dim;
	hipMalloc(&dimDev, 2*sizeof(unsigned int));
	hipMemcpy(dimDev, dim, 2*sizeof(unsigned int), hipMemcpyHostToDevice);

	// Tamanho de um caminho (soma das dimensões dos vértices)
	unsigned int dims = dim[0]+dim[1];
	// Threads por bloco (quantas amostras cabem num bloco)
	unsigned int threadsPerBlock = sharedMemPerBlock / (dims*sizeof(float));
	if (threadsPerBlock > maxThreadsPerBlock){
		// Número de threads ultrapassou o máximo permitido
		threadsPerBlock = maxThreadsPerBlock;
	}
	// Tamanho da memória compartilhada pelo bloco, 
	// utilizada para armazenar uma amostra por thread
	unsigned int sharedSize = threadsPerBlock * dims * sizeof(float);
	// Total de blocos
	unsigned int blocksPerGrid = train->total / threadsPerBlock;

	// Variável independente para associar à estimativa
	float *xDev;
	hipMalloc(&xDev, dim[0]*sizeof(float));

	// Cada bloco produz um par de somas parciais 
	// para cada dimensão da variável dependente
	float *yPart;
	float *yPartDev;
	hipHostMalloc(&yPart, 2*blocksPerGrid*dim[1]*sizeof(float));
	hipMalloc(&yPartDev, 2*blocksPerGrid*dim[1]*sizeof(float));

	// Variáveis para agregar o numerador e o denominador da fração
	// para cada dimensão da variável dependente
	float *numer;
	float *denom;
	hipHostMalloc(&numer, dim[1]*sizeof(float));
	hipHostMalloc(&denom, dim[1]*sizeof(float));

	// Vetor da estimativa da variável dependente
	float *y;
	hipHostMalloc(&y, dim[1]*sizeof(float));

	// Expressão envolvendo sigma no numerador do fator comum é constante
	float s = 2*pow(sigma,2);

	// Erro da estimativa
	float err = 0;

	// Índice de parcela
	unsigned int p;

	// Iterar em todo o conjunto de teste
	for (int i = 0; i < test->total; i++){
		// Copiar variável independente a ser estimada pela GPU
		hipMemcpy(xDev, &test->data.f[i*dims], dim[0]*sizeof(float), hipMemcpyHostToDevice);

		// Invocar kernel
		estimKernel<<<blocksPerGrid, threadsPerBlock, sharedSize>>>(trainDataDev, train->total, threadsPerBlock, dimDev, xDev, yPartDev, s);
		
		// Copiar parciais da estimativa geradas
		hipMemcpy(yPart, yPartDev, 2*blocksPerGrid*dim[1]*sizeof(float), hipMemcpyDeviceToHost);
	
		// Computar estimativa 
		for (unsigned int d = 0; d < dim[1]; d++){
			numer[d] = 0;
			denom[d] = 0;
			for (unsigned int b = 0; b < blocksPerGrid; b++){
				p = b*2*dim[1]+d;
				numer[d] += yPart[p];
				denom[d] += yPart[p+dim[1]];
				// Zerar parcial para a próxima amostra
				yPart[p] = 0;
				yPart[p+dim[1]] = 0;
			}
		}
		// Apagar parciais no dispositivo
		hipMemcpy(yPartDev, yPart, 2*blocksPerGrid*dim[1]*sizeof(float), hipMemcpyHostToDevice);

		// Vetor final da estimativa
		for (unsigned int d = 0; d < dim[1]; d++){
			y[d] = numer[d] / denom[d];
		}

		/*
		// Exibir valores da condição inicial
		printf("Condição inicial\n");
		for (int c = 0; c < dim[0]; c++){
			printf("%.6f\n", test->data.f[i*dims + c]);
		}
		// Exibir estimativa e valor observado
		printf("Valor Observado:\n");
		for (int c = 0; c < dim[1]; c++){
			printf("%.6f\n", test->data.f[i*dims + dim[0] + c]);
		}
		printf("Estimativa: \n");
		for (int c = 0; c < dim[1]; c++){
			printf("%.6f\n", y[c]);
		}
		*/
		// Erro da estimativa
		err = sqrt(dist(&test->data.f[i*dims], y, dim[1]));
		/*
		printf("Diferença: %f\n", err);
		printf("\n");
		*/
		// Erro acumulado (sem raiz)
		*errsum += err;
		// Mostrar progresso
		putchar('.');
		fflush(stdout);
		if ( (i+1) % 10 == 0 ){
			// Espaço a cada 10 pontos
			putchar(' ');
			fflush(stdout);
		}
	}
}

int main(int argc, char **argv){
	// Detectar GPU
	int deviceCount = 0;
	hipError_t error_id = hipGetDeviceCount(&deviceCount);
	if (error_id != hipSuccess){
		printf("Falha: hipGetDeviceCount devolveu %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
		exit(EXIT_FAILURE);
	}
	if (deviceCount == 0){
		printf("Nenhum dispositivo CUDA encontrado\n");
		exit(EXIT_FAILURE);
	}
	// Utilizar o primeiro dispositivo encontrado
	hipSetDevice(0);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	// Tamanho da memória compartilhada por bloco (em bytes)
	sharedMemPerBlock = deviceProp.sharedMemPerBlock;
	// Máximo de threads por bloco
	maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;

	// Mapeamento de memória entre o Host e a GPU
	hipSetDeviceFlags(hipDeviceMapHost);

	struct pathSet train, test;
	// Carregar arquivo das amostras de treinamento
	pathSetLoad(TRAIN, &train);

	// Arquivo de teste
	pathSetLoad(TEST, &test);

	printf("Conjunto de treinamento: %d amostras.\n", train.total);
	printf("Dimensões da variável independente: %d\n", train.dim[0]);
	printf("Dimensões da variável dependente:   %d\n", train.dim[1]);

	// Parâmetro sigma da regressão
	float sigma;
	// Erro acumulado
	float errsum;

	// Testar
	printf("Calculando estimativas para o conjunto teste (%d amostras).\n\n", test.total);
	puts("O ponto (.) representa a estimativa para cada amostra do conjunto de teste\n");

	// Parâmetro sigma (variância)
	sigma = 1.0/log(train.total);

	// Medir tempo e execução
	clock_t begin, end;

	// Paralelo
	begin = clock();
	errsum = 0;
	testarDev(&train, &test, sigma, &errsum);
	end = clock();
	//printf("\nTempo: %f segundos\n", (double)(end - begin) / CLOCKS_PER_SEC);
	printf("Erro médio: %f\n\n", errsum / (float)test.total);

	return 0;
}
