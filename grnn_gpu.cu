#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include "pathio.h"
#include "grnn_gpu.h"
#include <hip/hip_runtime.h>

// Arquivo das amostras
#define TRAIN "train.bin"
// Arquivos de teste
#define TEST "test.bin"

// Obter a quantidade de cuda cores por multiprocessador a partir do Compute Capability
inline int _ConvertSMVer2Cores(int major, int minor)
{
    // Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
    typedef struct
    {
        int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
        int Cores;
    } sSMtoCores;

    sSMtoCores nGpuArchCoresPerSM[] =
    {
        { 0x30, 192}, // Kepler Generation (SM 3.0) GK10x class
        { 0x32, 192}, // Kepler Generation (SM 3.2) GK10x class
        { 0x35, 192}, // Kepler Generation (SM 3.5) GK11x class
        { 0x37, 192}, // Kepler Generation (SM 3.7) GK21x class
        { 0x50, 128}, // Maxwell Generation (SM 5.0) GM10x class
        { 0x52, 128}, // Maxwell Generation (SM 5.2) GM20x class
        { 0x53, 128}, // Maxwell Generation (SM 5.3) GM20x class
        { 0x60, 64 }, // Pascal Generation (SM 6.0) GP100 class
        { 0x61, 128}, // Pascal Generation (SM 6.1) GP10x class
        { 0x62, 128}, // Pascal Generation (SM 6.2) GP10x class
        { 0x70, 64 }, // Volta Generation (SM 7.0) GV100 class

        {   -1, -1 }
    };

    int index = 0;

    while (nGpuArchCoresPerSM[index].SM != -1)
    {
        if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor))
        {
            return nGpuArchCoresPerSM[index].Cores;
        }

        index++;
    }

    // If we don't find the values, we default use the previous one to run properly
    printf("MapSMtoCores for SM %d.%d is undefined.  Default to use %d Cores/SM\n", major, minor, nGpuArchCoresPerSM[index-1].Cores);
    return nGpuArchCoresPerSM[index-1].Cores;
}

int main(int argc, char **argv){
	// Opções da linha de comando
	const char* outfile = NULL;
	float ss = 1;
	for(int i = 1; i < argc; i++){
		switch (argv[i][1]){
		case 'o':
			// Salvar resultado no arquivo indicado
			outfile = argv[i+1];
		break;
		case 's':
			// Escalar do parâmetro sigma
			ss = atof(argv[i+1]);
		break;
		}
	}

	// Identificar dispositivo
	init_gpu();

	// Hardware
	printf("Dispositivo: \"%s\"\n", deviceProp.name);
	printf("Capacidade: %d.%d\n", deviceProp.major, deviceProp.minor);
	printf("Multiprocessadores: %d\n", deviceProp.multiProcessorCount);
	printf("CUDA Cores / MP: %d\n", _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor));
	printf("Memória Global: %.0f MB\n", (float)deviceProp.totalGlobalMem/1048576.0f);
	// Driver e Runtime
	int driverVersion = 0, runtimeVersion = 0;
	hipDriverGetVersion(&driverVersion);
	hipRuntimeGetVersion(&runtimeVersion);
	printf("CUDA Driver: %d.%d\n", driverVersion/1000, (driverVersion%100)/10);
	printf("CUDA Runtime: %d.%d\n", runtimeVersion/1000, (runtimeVersion%100)/10);

	struct pathSet train, estim;
	// Carregar arquivo das amostras de treinamento
	pathSetLoad(TRAIN, &train);

	// Arquivo de teste
	pathSetLoad(TEST, &estim);

	printf("Conjunto de treinamento: %d amostras.\n", train.total);
	printf("Dimensões da variável independente: %d\n", train.dim[0]);
	printf("Dimensões da variável dependente:   %d\n", train.dim[1]);

	// Calcular o erro ou salvar um arquivo com o resultado
	printf("Estimando %d amostras de teste...\n", estim.total);
	// Soma dos erros das estimativas
	float errsum = 0;

	// Gerar estimativas
	estimar(&train, &estim, ss, &errsum);

	// Exibir erro médio
	printf("Erro médio: %f\n", errsum / (float)estim.total);

	// Salvar resultado no arquivo informado
	if (outfile != NULL ){
		pathSetSave(outfile, &estim);
		printf("Resultado salvo em %s\n", outfile);
	}

	return 0;
}
