#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include "pathio.h"
#include "grnn_gpu.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// Arquivo das amostras
#define TRAIN "train.bin"
// Arquivos de teste
#define TEST "test.bin"

int main(int argc, char **argv){
	// Opções da linha de comando
	const char* outfile = NULL;
	float ss = 1;
	for(int i = 1; i < argc; i++){
		switch (argv[i][1]){
		case 'o':
			// Salvar resultado no arquivo indicado
			outfile = argv[i+1];
		break;
		case 's':
			// Escalar do parâmetro sigma
			ss = atof(argv[i+1]);
		break;
		}
	}

	// Identificar dispositivo
	init_gpu();

	printf("Dispositivo: \"%s\"\n", deviceProp.name);
	int driverVersion = 0, runtimeVersion = 0;
	hipDriverGetVersion(&driverVersion);
	hipRuntimeGetVersion(&runtimeVersion);
	printf("CUDA Capability: %d.%d\n", deviceProp.major, deviceProp.minor);
	char msg[256];
	sprintf(msg, "Memória Global: %.0f MBytes (%llu bytes)\n", (float)deviceProp.totalGlobalMem/1048576.0f, (unsigned long long) deviceProp.totalGlobalMem);
	printf("%s", msg);
	printf("(%2d) Multiprocessors, (%3d) CUDA Cores/MP: %d CUDA Cores\n",
		deviceProp.multiProcessorCount,
		_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
		_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount);

	struct pathSet train, estim;
	// Carregar arquivo das amostras de treinamento
	pathSetLoad(TRAIN, &train);

	// Arquivo de teste
	pathSetLoad(TEST, &estim);

	printf("Conjunto de treinamento: %d amostras.\n", train.total);
	printf("Dimensões da variável independente: %d\n", train.dim[0]);
	printf("Dimensões da variável dependente:   %d\n", train.dim[1]);

	// Calcular o erro ou salvar um arquivo com o resultado
	printf("Estimando %d amostras de teste...\n", estim.total);
	// Soma dos erros das estimativas
	float errsum = 0;

	// Gerar estimativas
	estimar(&train, &estim, ss, &errsum);

	// Exibir erro médio
	printf("Erro médio: %f\n", errsum / (float)estim.total);

	// Salvar resultado no arquivo informado
	if (outfile != NULL ){
		pathSetSave(outfile, &estim);
		printf("Resultado salvo em %s\n", outfile);
	}

	return 0;
}
